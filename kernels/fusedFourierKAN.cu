#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>



//The function increment the value of out
//out should be initialized to zero beforehand
__global__ void ffkanGPUkernel( float* x, float* coeff, float* bias, int bs, int inputdim, int outputdim, int gridsize, float* out )
{
const int s_bs_out = outputdim;
const int s_bs_x = inputdim;

const int s_d_coeff= inputdim*outputdim*gridsize;
const int s_i_coeff = outputdim*gridsize;
const int s_o_coeff = gridsize;

for( int i = blockIdx.x ; i < bs ; i+=gridDim.x)
for( int j = 0 ; j < inputdim ; j++)
{
 float xx =  x[i*s_bs_x+j];
 float c0 = cosf(xx); 
 float s0 = sinf(xx);
for( int l = threadIdx.x ; l < outputdim ; l+=blockDim.x)
{
 float ckm = 1.0f;
 float skm = 0.0f;
for( int k = 1 ; k < gridsize+1 ; k++)
{
 //For better performance We use trig formula to compute ck,sk from ck-1, sk-1, cos(xx),sin(xx)
 //But this form is better to obtain the bacwkard pass
 //float c = cos(k*xx); 
 //float s = sin(k*xx);
 float c = ckm*c0-skm*s0;
 float s = skm*c0+ckm*s0;
 ckm = c;
 skm = s;
 out[i*s_bs_out+l] += coeff[s_d_coeff*0 + s_i_coeff*j + s_o_coeff*l + k-1] * c;
 out[i*s_bs_out+l] += coeff[s_d_coeff*1 + s_i_coeff*j + s_o_coeff*l + k-1] * s;
}

}
}

for( int i = blockIdx.x ; i < bs ; i+=gridDim.x)
for( int l = threadIdx.x ; l < outputdim ; l+=blockDim.x)
    out[i*s_bs_out+l] += bias[l];

}

void ffkanGPU( float* x, float* coeff, float* bias, int bs, int inputdim, int outputdim, int gridsize, float* out )
{
dim3 blockDims(32);
dim3 gridDims( min(bs,1024) );
ffkanGPUkernel<<<gridDims,blockDims>>>(x,coeff,bias,bs,inputdim,outputdim,gridsize,out);
}

//The function should not use the value of out and doesn't
 template< bool partialBias, bool partialx, bool partialcoeff>
__global__ void ffkanGPUkernel_b(float *x, float *xb, float *coeff, float *coeffb, float *bias, 
        float *biasb, int bs, int inputdim, int outputdim, int gridsize, float
        *out, float *outb) {
    const int s_bs_out = outputdim;
    const int s_bs_x = inputdim;
    const int s_d_coeff = inputdim*outputdim*gridsize;
    const int s_i_coeff = outputdim*gridsize;
    const int s_o_coeff = gridsize;

    //These loops should be iterated backward according to autodiff, but if we assume (even if not true) commutativity and associativity of floating point addition 
    //we can iterate them in the nicer looking normal order
    if( partialBias )
    for( int i = 0 ; i < bs ; i++)
        for( int l = threadIdx.x ; l < outputdim ; l+=blockDim.x )
            biasb[l] += outb[i*s_bs_out + l];

    //These loops should be iterated backward according to autodiff, but if we assume (even if not true) commutativity and associativity of floating point addition 
    //we can iterate them in the nicer looking normal order
    for( int  i = blockIdx.y ; i < bs ; i+=gridDim.y)
    for( int j = threadIdx.x ; j < inputdim ; j+=blockDim.x)
    {
    float xx =  x[i*s_bs_x+j];
    float c0 = cosf(xx); 
    float s0 = sinf(xx);
    for( int l = blockIdx.x ; l < outputdim ; l+=gridDim.x)
    {
    float ckm = 1.0f;
    float skm = 0.0f;
    for( int k = 1 ; k < gridsize+1 ; k++)
                   {
                    //float xx = x[i*s_bs_x + j];
                    float xxb = 0.0;
                    //For better performance We use trig formula to compute ck,sk from ck-1, sk-1, cos(xx),sin(xx)
                    //But this form is better to obtain the bacwkard pass
                    //float c = cos(k*xx);
                    //float s = sin(k*xx);
                    float c = ckm*c0-skm*s0;
                    float s = skm*c0+ckm*s0;
                    ckm = c;
                    skm = s;
                    float cb;
                    float sb;
                    if( partialcoeff)
                    {
                        coeffb[s_d_coeff*1 + s_i_coeff*j + s_o_coeff*l + k-1] += s*outb[i*s_bs_out+l];
                        coeffb[s_d_coeff*0 + s_i_coeff*j + s_o_coeff*l + k-1] += c*outb[i*s_bs_out+l];
                    }
                    if( partialx)
                    {
                    sb = coeff[s_d_coeff*1 + s_i_coeff*j + s_o_coeff*l + k-1] * outb[i*s_bs_out+l];
                    cb = coeff[s_d_coeff*0 + s_i_coeff*j + s_o_coeff*l + k-1]*outb[i*s_bs_out+l];
                    xxb = k*c*sb - k*s*cb;
                    xb[i*s_bs_x + j] += xxb;
                    }
                    
                   }
    }
    }
}

void ffkanGPU_b(float *x, float *xb, float *coeff, float *coeffb, float *bias, 
        float *biasb, int bs, int inputdim, int outputdim, int gridsize, float
        *out, float *outb)
{
//Compute bias gradient
ffkanGPUkernel_b<true,false,false><<<1, min(outputdim,1024) >>>(x,xb,coeff,coeffb,bias,biasb,bs,inputdim,outputdim,gridsize,out,outb);
dim3 gridDimX(1,min(bs,1024),1);
//Compute x gradient
ffkanGPUkernel_b<false,true,false><<<gridDimX, min(inputdim,1024) >>>(x,xb,coeff,coeffb,bias,biasb,bs,inputdim,outputdim,gridsize,out,outb);
//Compute coeff gradient
dim3 gridDimCoeff(min(outputdim,1024),1,1);
ffkanGPUkernel_b<false,false,true><<<gridDimCoeff, min(inputdim,1024) >>>(x,xb,coeff,coeffb,bias,biasb,bs,inputdim,outputdim,gridsize,out,outb);


}